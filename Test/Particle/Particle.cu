#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "glm.hpp"
// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <>


__global__ void calcParticlePos(float* particlePos, float* inc, float* result,const float incMultiplier)
{
    int it = threadIdx.x;
    float incResult  = (particlePos[it] - inc[it]) * incMultiplier;
    result[it] = particlePos[it] + 1.f * (incResult - particlePos[it]);

}

__global__ void incVelocity(float* currentVec, float* inc, float* result)
{
    int it = threadIdx.x;
    result[it] = currentVec[it] + inc[it];
}

__global__ void rotate(float* current,float* inc, float* result)
{
    int it = threadIdx.x;
    result[it] = current[it] + inc[it];
}


void rotate(const glm::vec3& particlePos,const glm::vec3& inc,float incMultiplier)
{
    calcParticlePos<<<>>>;
}